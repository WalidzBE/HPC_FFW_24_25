#include "hip/hip_runtime.h"
/**********************************************************************
 * Gaussian Blur Image Processing with CUDA
 * Applies a 3x3 Gaussian blur to RGB images
 *********************************************************************/
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <iostream>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32  // Default size for thread blocks
#endif

// Gaussian kernel weights stored in constant GPU memory
__constant__ int gaussian_kernel_weights[9] = {1, 2, 1, 
                                               3, 4, 3, 
                                               1, 2, 1};

// Mirror device to handle boundaries
__device__ int mirror(int x, int max) {
    if (x < 0) return -x;            // mirror -1 to 1
    if (x >= max) return 2 * max - x - 2; // mirror N to N-2
    return x;
}
    
/* --- CUDA kernel for applying 3x3 Gaussian blur ------------------- */
__global__ void applyGaussianBlur(const uchar3 *input_image, uchar3 *output_image,
                                 int image_width, int image_height, 
                                 size_t memory_pitch, int row_stride)
{
    // Shared memory for storing image tile with halo pixels
    __shared__ unsigned char shared_tile[(BLOCK_SIZE+2)*(BLOCK_SIZE+2)*3];

    // Calculate global pixel coordinates
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate local coordinates within shared memory (including halo)
    int local_x = threadIdx.x + 1;
    int local_y = threadIdx.y + 1;

    // Fill shared memory tile using mirrored boundary pixels
    for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
            int src_x = mirror(global_x + dx, image_width);
            int src_y = mirror(global_y + dy, image_height);

            int tile_x = local_x + dx;
            int tile_y = local_y + dy;

            if (tile_x >= 0 && tile_x < BLOCK_SIZE + 2 && tile_y >= 0 && tile_y < BLOCK_SIZE + 2) {
                *(uchar3*)&shared_tile[(tile_y * (BLOCK_SIZE + 2) + tile_x) * 3] =
                    input_image[src_y * row_stride + src_x];
            }
        }
    }

    // Ensure all threads have finished copying their portion of the image into shared memory and halo Pixels are visible to all other threads before the computation begins
    __syncthreads();
    
    // Apply Gaussian blur if within image bounds
    if (global_x < image_width && global_y < image_height) 
    {
        // Initialize sum for RGB channels
        int3 channel_sums = {0, 0, 0};
        
        // Apply 3x3 kernel weights to surrounding pixels
        for (int kernel_y = -1; kernel_y <= 1; ++kernel_y)
        {
            for (int kernel_x = -1; kernel_x <= 1; ++kernel_x) 
            {
                uchar3 pixel = *(uchar3*)&shared_tile[((local_y+kernel_y)*(BLOCK_SIZE+2)+local_x+kernel_x)*3];
                int weight = gaussian_kernel_weights[(kernel_y+1)*3 + (kernel_x+1)];
                channel_sums.x += weight * pixel.x;
                channel_sums.y += weight * pixel.y;
                channel_sums.z += weight * pixel.z;
            }
        }
        
        // Normalize and store result
        uchar3 result_pixel;
        const int normalization_factor = 18;
        const int rounding_offset = 9;
        result_pixel.x = static_cast<unsigned char>((channel_sums.x + rounding_offset) / normalization_factor);
        result_pixel.y = static_cast<unsigned char>((channel_sums.y + rounding_offset) / normalization_factor);
        result_pixel.z = static_cast<unsigned char>((channel_sums.z + rounding_offset) / normalization_factor);
        output_image[global_y*row_stride + global_x] = result_pixel;
    }
}

/* --------------------------- Main Program --------------------------- */
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        std::cerr << "[CUDA ERROR] " << hipGetErrorString(err) \
                  << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

int main(int argc, char **argv)
{
    // Check command line arguments
    if (argc != 4) 
    {
        std::cerr << "Usage: program_name input_image.png output_image.png log_label\n";
        return 1;
    }


    // Load input image
    cv::Mat input_image = cv::imread(argv[1], cv::IMREAD_COLOR);
    if (input_image.empty()) 
    {
        std::cerr << "Error: Could not load input image\n"; 
        return 1;
    }

    // Get image dimensions
    int image_width = input_image.cols;
    int image_height = input_image.rows;
    size_t memory_pitch = static_cast<size_t>(image_width) * sizeof(uchar3);

    // Allocate GPU memory
    uchar3 *device_input_image, *device_output_image;
    CUDA_CHECK(hipMalloc(&device_input_image, image_height * memory_pitch));
    CUDA_CHECK(hipMalloc(&device_output_image, image_height * memory_pitch));
    CUDA_CHECK(hipMemcpy(device_input_image, input_image.ptr(), image_height * memory_pitch, hipMemcpyHostToDevice));

    std::cout << "[DEBUG] Copied device data to GPU" << std::endl;

    // Setup CUDA grid and block dimensions
    dim3 thread_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_dimensions((image_width + BLOCK_SIZE-1)/BLOCK_SIZE, (image_height + BLOCK_SIZE-1)/BLOCK_SIZE);
    
    std::cout << "[DEBUG] Launching kernel with block (" << thread_block.x << ", " << thread_block.y
                  << ") and grid (" << grid_dimensions.x << ", " << grid_dimensions.y << ")" << std::endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Apply Gaussian blur
    applyGaussianBlur<<<grid_dimensions, thread_block>>>(device_input_image, device_output_image, 
                                                       image_width, image_height, memory_pitch, image_width);

    // Check kernel launch error
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "[LAUNCH ERROR] " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Now check runtime errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "[RUNTIME ERROR] " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    std::cout << "[DEBUG] Kernel completed" << std::endl;

    // Copy result back to host and save
    cv::Mat output_image(input_image.size(), input_image.type());
    CUDA_CHECK(hipMemcpy(output_image.ptr(), device_output_image, image_height * memory_pitch, hipMemcpyDeviceToHost));

    std::cout << "[DEBUG] Copied device data back to host" << std::endl;

    // Check output file extension before saving
    std::string out_file = argv[2];
    std::string ext = out_file.substr(out_file.find_last_of('.') + 1);
    std::vector<std::string> supported_ext = {"png", "jpg", "jpeg", "bmp", "tiff", "tif"};

    if (out_file == "null") {
        std::cout << "Output file is 'null', skipping save." << std::endl;
    }
    else if (std::find(supported_ext.begin(), supported_ext.end(), ext) == supported_ext.end()) {
        std::cerr << "Error: Unsupported output file extension: " << ext << std::endl;
        std::cerr << "Supported extensions: png, jpg, jpeg, bmp, tiff, tif" << std::endl;
    }
    else {
        cv::imwrite(out_file, output_image);
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Block size: %d x %d | Time: %f ms\n", BLOCK_SIZE, BLOCK_SIZE, milliseconds);


    // Append to CSV
    FILE* f = fopen("results.csv", "a");
    if (f != NULL) {
        fprintf(f, "BASIC,%s,%d,%f\n", argv[3], BLOCK_SIZE, milliseconds);
        fclose(f);
    } else {
        fprintf(stderr, "Failed to write to results.csv\n");
    }

    // Cleanup
    hipFree(device_input_image); 
    hipFree(device_output_image);
    return 0;
}